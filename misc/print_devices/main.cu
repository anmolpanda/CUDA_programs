#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>

int main() {
	int deviceCount {};
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0) {
		std::cerr << "Error: no devices supporting CUDA.\n";
		return -1;
	}

	for (int dev = 0; dev < deviceCount; ++dev) {
		hipSetDevice(dev);
		hipDeviceProp_t devProps;
		if(hipGetDeviceProperties(&devProps, dev) == 0){
			std::cout << "Device       " << dev << "\n";
			std::cout << "GPU name:    " << devProps.name << "\n";
			std::cout << "global mem:  " << (unsigned int)devProps.totalGlobalMem/(1024*1024) << " MBytes" << "\n";
			std::cout << "compute cap: " << (int)devProps.major << "." << (int)devProps.minor << "\n";
			std::cout << "clock:       " << (int)devProps.clockRate << " kHz\n";
		}
	}
	return 0;
}