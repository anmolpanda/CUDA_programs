#include "hip/hip_runtime.h"
#include "spdia.h"


__global__ void sparseDiagMatrixMultKernel(int *d_SDM, int *d_indices, int *d_inVec, 
							int *d_outVec, const int vecSize, const int indicesSize) {
	// syntactic sugar - splitting shared memory block into 
	// 					 2 shared memory arrays
	extern __shared__ int shared[];
  	int *ds_inVec = shared;
  	int *ds_outVec = &shared[blockDim.x];

	int tx = threadIdx.x;
	int tid = tx + blockIdx.x * blockDim.x;


	if (tid < vecSize) {
		ds_outVec[tx] = 0;
		for(int tileIndex = 0; tileIndex != (vecSize - 1) / TILE_WIDTH + 1; ++tileIndex) {
			ds_inVec[tx] = d_inVec[tx + tileIndex * TILE_WIDTH];
			__syncthreads();
			printf("Thread %d, tileIndex:: %d,  val: %d\n", 
					tid, tileIndex, ds_inVec[tx]);

			for (int i = 0; i != indicesSize; ++i){ // oops -> complexity
				//printf("Thread %d, i: %d,  val: %d\n", tx, i, d_SDM[tx + i * vecSize]);
				int diagInd = d_indices[i]; 
				if(-tid + TILE_WIDTH * tileIndex <= diagInd 
					&& diagInd < TILE_WIDTH * (tileIndex + 1) - tid) {
					printf("Thread: %d, tileIndex: %d, diagInd: %d, ds_inVec[diagInd + tid]: %d, d_SDM[i * vecSize + tid]: %d\n", 
							tid, tileIndex, diagInd, ds_inVec[diagInd + tid - tileIndex * TILE_WIDTH], 
							d_SDM[i * vecSize + tid]);
					ds_outVec[tx] += (d_SDM[i * vecSize + tid] * ds_inVec[diagInd + tid - tileIndex * TILE_WIDTH]);
				}
			}
			__syncthreads();
		}



		d_outVec[tid] = ds_outVec[tx];
	}
}

void sparseDiagMatrixMult(std::vector<int> &h_SDM, std::vector<int> &h_indices, 
	std::vector<int> &h_inVec, std::vector<int> &h_outVec) 
{
	const int vecSize = h_inVec.size();
	int *d_SDM;
	int *d_indices;
	int *d_inVec;
	int *d_outVec;

	const int SDM_BYTE_SIZE = h_SDM.size() * sizeof(int);
	const int VEC_BYTE_SIZE = h_inVec.size() * sizeof(int);
	const int VEC_INDICES_BYTE_SIZE = h_indices.size() * sizeof(int);
	
	hipMalloc((void**)& d_SDM, SDM_BYTE_SIZE);
	hipMalloc((void**)& d_inVec, VEC_BYTE_SIZE);
	hipMalloc((void**)& d_outVec, VEC_BYTE_SIZE);
	hipMalloc((void**)& d_indices, VEC_INDICES_BYTE_SIZE);
	
	hipMemcpy(d_SDM, h_SDM.data(), SDM_BYTE_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(d_inVec, h_inVec.data(), VEC_BYTE_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(d_indices, h_indices.data(), VEC_INDICES_BYTE_SIZE, hipMemcpyHostToDevice);

	dim3 grid((vecSize - 1) / TILE_WIDTH + 1);
	dim3 block(TILE_WIDTH);
	std::cout << "Threads per block: " << TILE_WIDTH << " blocks per grid: " 
			  <<  (vecSize - 1) / TILE_WIDTH + 1 << "\n";
	const int SH_MEM_BYTE_SIZE = 2 * TILE_WIDTH * sizeof(int);
	sparseDiagMatrixMultKernel<<<grid, block, SH_MEM_BYTE_SIZE>>>(d_SDM,
				d_indices, d_inVec, d_outVec, vecSize, h_indices.size());

	hipMemcpy(h_outVec.data(), d_outVec, VEC_BYTE_SIZE, hipMemcpyDeviceToHost);

	hipFree(d_SDM);
	hipFree(d_inVec);
	hipFree(d_outVec);
	hipFree(d_indices);
}